#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include<stdlib.h>
#include<iostream>
#include<hip/hip_runtime.h>

using namespace std ;

#define THR 512 // 2^9
#define BLK 2048 // 2^11
#define NUM BLK*THR // 2^20

void print_elapsed(clock_t start, clock_t stop)
{
	double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC ; // elapsed = how much time goes on
	cout << "Elapsed time : " << elapsed << "sec" << endl ; // print time (sec) ;
}

__global__ void oddeven(int *arr, int cond)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x ;
	int temp ;
	if (index >= (NUM / 2) - 1 && cond % 2 != 0) return ;

	if( cond % 2 == 0 ) // if cond is even
	{
		if( arr[index * 2] > arr[index * 2 + 1] ) // compare arr[0]<->arr[1], arr[2]<->arr[3], ... // (0, 2, 4, ... even)
		{
			temp = arr[index * 2] ;
			arr[index * 2] = arr[index * 2 + 1] ;
			arr[index * 2 + 1] = temp ; // if arr[0] > arr[1], arr[2] > arr[3], ... change each other
		}
	}
	else // if cond is odd
	{
		if( arr[index * 2 + 1] > arr[index * 2 + 2] ) // compare arr[1]<->arr[2], arr[3]<->arr[4], ... (1, 3, 5, ... even)
		{
			temp = arr[index * 2 + 1] ;
			arr[index * 2 + 1] = arr[index * 2 + 2] ;
			arr[index * 2 + 2] = temp ; // if arr[1] > arr[2], arr[3] > arr[4], ... change each other
		}
	}
}

int main()
{
	clock_t start, stop ;

	int *values ;
	int *deva_values ;
	int i ;
	int size = sizeof(int) * NUM ;

	values = (int *)malloc(size) ; // dynamic memory allocation

	hipMalloc((void **)&deva_values, size) ; // using cuda memory allocation

	srand(unsigned(time(NULL))) ; // it occurs each random number at every each time

	for (i = 0; i < NUM; i++)
	{
		values[i] = rand(); // put random number into array
	}

	hipMemcpy(deva_values, values, size, hipMemcpyHostToDevice) ; // using cuda copy value -> dev_values (size) // Host to Device

	start = clock() ; // time start

	for (i = 0; i < NUM; i++)
	{
		oddeven << <NUM / 1024, 512 >> >(deva_values, i) ; //512 threads per block and total N/2/512 blocks
	}

	stop = clock() ; // time stop

	hipMemcpy(values, deva_values, size, hipMemcpyDeviceToHost) ; // using cuda copy dev_values -> values (size) // Device to Host

	cout << "12141680 GPU OddEven-Sort\n" << endl ;

	print_elapsed(start, stop) ; // print elapsed time

	cout << endl ;

	for (i = 0; i<NUM; i++)
	{
		printf("%d ", values[i]) ; // print sorted number
	}

	hipFree(deva_values) ; // free memory

	delete values ;

	return 0;
}