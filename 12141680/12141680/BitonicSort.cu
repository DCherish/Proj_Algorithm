#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include<stdlib.h>
#include<iostream>
#include<hip/hip_runtime.h>

using namespace std ;

#define THREADS 512 // 2^9
#define BLOCKS 2048 // 2^11
#define NUM_VALS THREADS*BLOCKS // 2^20

void print_elapsed(clock_t start, clock_t stop)
{
	double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC ; // elapsed = how much time goes on
	cout << "Elapsed time : " << elapsed << "sec" << endl ; // print time (sec) ;
}

int random_int()
{
	return (int)rand() ; // return random number
}

void array_print(int *arr, int length)
{
	for (int i = 0; i < length; ++i)
	{
		cout << arr[i] << " " ; // print sorted number
	}

	cout << endl ;
}

void array_fill(int *arr, int length)
{
	srand(unsigned(time(NULL))) ; // it occurs each random number at every each time

	for( int i = 0 ; i < length ; ++i )
	{
		arr[i] = random_int() ; // put random number into array
	}
}

__global__ void bitonic_sort_step(int *dev_values, int j, int k) // bitonicSorting
{
	unsigned int i, itj ; // Sorting partners, i and itj and it can't be negative number
	i = threadIdx.x + blockDim.x * blockIdx.x ;
	itj = i^j ;

	if( (itj) > i )
	{
		if( (i&k) == 0 ) // Sort ascending
		{
			if( dev_values[i] > dev_values[itj] ) // if v[i] > v[itj]
			{
				int temp = dev_values[i] ;
				dev_values[i] = dev_values[itj] ;
				dev_values[itj] = temp ; // exchange(i,itj) 
			}
		}

		if( (i&k) != 0 )// Sort descending 
		{
			if( dev_values[i] < dev_values[itj] ) // if v[i] < v[itj]
			{
				int temp = dev_values[i] ;
				dev_values[i] = dev_values[itj] ;
				dev_values[itj] = temp ; // exchange(i,itj)
			}
		}
	}
}

void bitonic_sort(int *values) // bitonicSort
{
	int *dev_values ;
	size_t size = NUM_VALS * sizeof(int) ;

	hipMalloc((void**)&dev_values, size) ; // using cuda memory allocation
	hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice) ; // using cuda copy values -> dev_values (size) // Host to Device

	dim3 blocks(BLOCKS, 1) ; // Number of blocks  
	dim3 threads(THREADS, 1) ; // Number of threads

	int j, k ; // in device, do sort

	for( k = 2 ; k <= NUM_VALS ; k <<= 1 )
	{
		for( j = k >> 1 ; j > 0 ; j = j >> 1 )
		{
			bitonic_sort_step << <blocks, threads >> >(dev_values, j, k) ;
		}
	}
	hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost) ; // using cuda copy dev_values -> values (size) // Device to Host
	hipFree(dev_values) ; // free memory
}

int main(void)
{
	clock_t start, stop ;

	int *values = (int*)malloc(NUM_VALS * sizeof(int)) ; // dynamic memory allocation

	array_fill(values, NUM_VALS) ; // execute array fill (put random number into array)

	start = clock() ; // time start
	bitonic_sort(values) ; // execute bitonic sort
	stop = clock() ; // time stop

	cout << "12141680 GPU Binotic-Sort\n" << endl ;

	print_elapsed(start, stop) ; // print elapsed time

	cout << endl ;

	array_print(values, NUM_VALS) ; // print sorted number

	delete values;
}